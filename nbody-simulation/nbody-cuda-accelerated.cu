#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
//#include "timer.h"
//include "check.h"
#include <chrono>

#define nthreads 32
#define SOFTENING 1e-9f

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

typedef struct { float x, y, z, vx, vy, vz; } Body;

void randomizeBodies(float *data, int n) { // this should remain a host function
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

/*
 * This function calculates the gravitational impact of all bodies in the system
 * on all others, but does not update their positions.
 */

__global__
void bodyForce(Body *p, float dt, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i<n) 
  {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
    for(int tl = 0; tl <gridDim.x; tl++)
    {
      __shared__ float3 position_in_shared_mem[nthreads]; // usign shared memory on gpu
      float position_in_x = p[tl * blockDim.x + threadIdx.x].x;
      float position_in_y = p[tl * blockDim.x + threadIdx.x].y;
      float position_in_z = p[tl * blockDim.x + threadIdx.x].z;
      position_in_shared_mem[threadIdx.x] = make_float3(position_in_x, position_in_y, position_in_z);
      __syncthreads();
      for(int j=0; j<nthreads; j++)
      {
        float dx = position_in_shared_mem[j].x - p[i].x;
	float dy = position_in_shared_mem[j].y - p[i].y;
	float dz = position_in_shared_mem[j].z - p[i].z;
	float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
        float invDist = rsqrtf(distSqr);
        float invDist3 = invDist * invDist * invDist;

	Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
      }
      __syncthreads();
     }
     p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
  }
}

int main(const int argc, const char** argv) {


  int nBodies = 2<<11;
  int salt = 0;
  if (argc > 1) nBodies = 2<<atoi(argv[1]);

  /*
   * This salt is for assessment reasons. Tampering with it will result in automatic failure.
   */

  if (argc > 2) salt = atoi(argv[2]);

  const float dt = 0.01f; // time step
  const int nIters = 10;  // simulation iterations

  int nblock = (nBodies + nthreads - 1)/nthreads;

  hipDeviceProp_t props;
  int deviceId;
  hipGetDevice(&deviceId); 
  hipGetDeviceProperties(&props, deviceId);
  int computeCapabilityMajor = props.major;
  int computeCapabilityMinor = props.minor;
  int multiProcessorCount = props.multiProcessorCount;
  int warpSize = props.warpSize;
  int bytes = nBodies * sizeof(Body);
  float *buf;

  //buf = (float *)malloc(bytes);
  hipMallocManaged(&buf,bytes);
  hipMemPrefetchAsync(buf,bytes,deviceId);

  Body *p = (Body*)buf;


  randomizeBodies(buf, 6 * nBodies); // Init pos / vel data

  double totalTime = 0.0;


  printf("Device ID: %d\nNumber of SMs: %d\nCompute Capability Major: %d\nCompute Capability Minor: %d\nWarp Size: %d\n", deviceId, multiProcessorCount, computeCapabilityMajor, computeCapabilityMinor, warpSize);
  /*
   * This simulation will run for 10 cycles of time, calculating gravitational
   * interaction amongst bodies, and adjusting their positions to reflect.
   */
 
  /*******************************************************************/
  for (int iter = 0; iter < nIters; iter++) {
 auto start=std::chrono::high_resolution_clock::now();	  
    
 bodyForce<<<nblock,nthreads>>>(p,dt,nBodies);
  /*
   * This position integration cannot occur until this round of `bodyForce` has completed.
   * Also, the next round of `bodyForce` cannot begin until the integration is complete.
   */

    hipDeviceSynchronize();
    for (int i = 0 ; i < nBodies; i++) { // integrate position
      p[i].x += p[i].vx*dt;
      p[i].y += p[i].vy*dt;
      p[i].z += p[i].vz*dt;
    }

  /*******************************************************************/
    //const double tElapsed = GetTimer() / 1000.0;
    //totalTime += tElapsed;
    auto finish = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> tElapsed = finish - start;
    totalTime += tElapsed.count();
  }

  hipDeviceSynchronize();
  double avgTime = totalTime / (double)(nIters);
  float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

#ifdef ASSESS
  checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
  //checkAccuracy(buf, nBodies);
  printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
  salt += 1;
#endif
  /*******************************************************************/


  //free(buf);
  hipFree(buf);
}

